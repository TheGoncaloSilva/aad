#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */

__global__ void helloFromGPU()
{
    printf("Hello World from GPU:(%d,%d,%d)-(%d,%d,%d)!\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
    // We can see that only one block was used with a linear line of threads only. 
    //  The execution is conducted in order
}

int main(int argc, char **argv)
{

    printf("Hello World from CPU!\n");

    // send data to a computational unit
    // 1, 10 are the launching geometry with just 1 thread block with 10 threads
    //helloFromGPU<<<1, 10>>>();
    dim3 grid, block;
    grid = {2,5,1}; // 10 blocks
    block = {1,4,1};    // 4 threads in each block
    helloFromGPU<<<grid, block>>>();
    CHECK(hipDeviceReset());
    return 0;
}


